#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h> // include CUDA runtime API
#include <chrono>         // include chrono for CPU timing

// GPU Kernel
__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

// CPU Function
void saxpy_cpu(int n, float a, float *x, float *y)
{
    for (int i = 0; i < n; i++) {
        y[i] = a * x[i] + y[i];
    }
}

int main(void)
{
    int start_power = 15;
    int end_power = 25;

    printf("MatrixSize,HostMalloc_ms,DeviceMalloc_ms,MemcpyHtoD_ms,Kernel_ms,MemcpyDtoH_ms,FreeHost_ms,FreeDevice_ms,TotalMeasured_ms\n");

    for (int p = start_power; p <= end_power; p++) {
        int N = 1 << p;
        float *x, *y, *d_x, *d_y;

        // Create CUDA events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        float host_malloc_time = 0.0f;
        float device_malloc_time = 0.0f;
        float memcpy_h2d_time = 0.0f;
        float kernel_time = 0.0f;
        float memcpy_d2h_time = 0.0f;
        float host_free_time = 0.0f;
        float device_free_time = 0.0f;

        // Host malloc timing
        hipEventRecord(start);
        x = (float*)malloc(N * sizeof(float));
        y = (float*)malloc(N * sizeof(float));
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&host_malloc_time, start, stop);

        // Device malloc timing
        hipEventRecord(start);
        hipMalloc(&d_x, N * sizeof(float));
        hipMalloc(&d_y, N * sizeof(float));
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&device_malloc_time, start, stop);

        // Initialize host arrays (does not time it separately)
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        // Host to device memcpy timing
        hipEventRecord(start);
        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&memcpy_h2d_time, start, stop);

        // Kernel execution timing
        hipEventRecord(start);
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&kernel_time, start, stop);

        // Device to host memcpy timing
        hipEventRecord(start);
        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&memcpy_d2h_time, start, stop);

        // Host free timing
        hipEventRecord(start);
        free(x);
        free(y);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&host_free_time, start, stop);

        // Device free timing
        hipEventRecord(start);
        hipFree(d_x);
        hipFree(d_y);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&device_free_time, start, stop);

        // Total
        float total_measured_time = host_malloc_time + device_malloc_time + memcpy_h2d_time + kernel_time + memcpy_d2h_time + host_free_time + device_free_time;

        // Print results
        printf("[GPU] %d,%.5f,%.5f,%.5f,%.5f,%.5f,%.5f,%.5f,%.5f\n", 
            N,
            host_malloc_time,
            device_malloc_time,
            memcpy_h2d_time,
            kernel_time,
            memcpy_d2h_time,
            host_free_time,
            device_free_time,
            total_measured_time
        );

        // Destroy events
        hipEventDestroy(start);
        hipEventDestroy(stop);

        // === Now benchmark CPU ===
        float *x_cpu, *y_cpu;

        double host_malloc_time_cpu = 0.0;
        double computation_time_cpu = 0.0;
        double host_free_time_cpu = 0.0;

        auto cpu_start = std::chrono::high_resolution_clock::now();
        x_cpu = (float*)malloc(N * sizeof(float));
        y_cpu = (float*)malloc(N * sizeof(float));
        auto cpu_stop = std::chrono::high_resolution_clock::now();
        host_malloc_time_cpu = std::chrono::duration<double, std::milli>(cpu_stop - cpu_start).count();

        for (int i = 0; i < N; i++) {
            x_cpu[i] = 1.0f;
            y_cpu[i] = 2.0f;
        }

        cpu_start = std::chrono::high_resolution_clock::now();
        saxpy_cpu(N, 2.0f, x_cpu, y_cpu);
        cpu_stop = std::chrono::high_resolution_clock::now();
        computation_time_cpu = std::chrono::duration<double, std::milli>(cpu_stop - cpu_start).count();

        cpu_start = std::chrono::high_resolution_clock::now();
        free(x_cpu);
        free(y_cpu);
        cpu_stop = std::chrono::high_resolution_clock::now();
        host_free_time_cpu = std::chrono::duration<double, std::milli>(cpu_stop - cpu_start).count();

        double total_cpu_time = host_malloc_time_cpu + computation_time_cpu + host_free_time_cpu;

        printf("[CPU] %d, %.5f, %.5f, %.5f, %.5f\n", 
            N, host_malloc_time_cpu, computation_time_cpu, host_free_time_cpu, total_cpu_time);
    }

    return 0;
}
