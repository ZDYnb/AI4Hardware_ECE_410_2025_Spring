#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h> //include CUDA runtime API to use GPU functions

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}// that define a CUDA kernel runs on GPU called saxpy

int main(void)
{
    int start_power = 15;
    int end_power = 25;

    // Print CSV headers
    printf("MatrixSize,TotalTime_ms,KernelTime_ms\n");

    for (int p = start_power; p <= end_power; p++) {
        int N = 1 << p;
        float *x, *y, *d_x, *d_y;

        // Timing
        hipEvent_t start_total, stop_total, start_kernel, stop_kernel;
        hipEventCreate(&start_total);
        hipEventCreate(&stop_total);
        hipEventCreate(&start_kernel);
        hipEventCreate(&stop_kernel);

        // Start total timer
        hipEventRecord(start_total);

        // Allocate host memory
        x = (float*)malloc(N * sizeof(float));
        y = (float*)malloc(N * sizeof(float));

        // Allocate device memory
        hipMalloc(&d_x, N * sizeof(float));
        hipMalloc(&d_y, N * sizeof(float));

        // Initialize host arrays
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        // Copy data from host to device
        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

        // Start kernel-only timer
        hipEventRecord(start_kernel);

        // Launch kernel
        saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y);

        // Stop kernel-only timer
        hipEventRecord(stop_kernel);

        // Copy result back to host
        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

        // Stop total timer
        hipEventRecord(stop_total);

        // Wait for events to complete
        hipEventSynchronize(stop_total);
        hipEventSynchronize(stop_kernel);

        // Calculate elapsed time
        float total_time_ms = 0.0f;
        float kernel_time_ms = 0.0f;
        hipEventElapsedTime(&total_time_ms, start_total, stop_total);
        hipEventElapsedTime(&kernel_time_ms, start_kernel, stop_kernel);

        // Print results in CSV format
        printf("%d,%.5f,%.5f\n", N, total_time_ms, kernel_time_ms);

        // Free memory
        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);

        // Destroy events
        hipEventDestroy(start_total);
        hipEventDestroy(stop_total);
        hipEventDestroy(start_kernel);
        hipEventDestroy(stop_kernel);
    }

    return 0;
}
