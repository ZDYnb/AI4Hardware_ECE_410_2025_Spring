#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

// GPU device function: iterative Fibonacci
__device__ unsigned long long fibonacci_device(int n)
{
    if (n <= 1) return n;
    unsigned long long a = 0, b = 1, c;
    for (int i = 2; i <= n; ++i) {
        c = a + b;
        a = b;
        b = c;
    }
    return b;
}

// GPU Kernel
__global__
void fibonacci_kernel(int *input, unsigned long long *output, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = fibonacci_device(input[idx]);
    }
}

// CPU Fibonacci function
unsigned long long fibonacci_cpu(int n)
{
    if (n <= 1) return n;
    unsigned long long a = 0, b = 1, c;
    for (int i = 2; i <= n; ++i) {
        c = a + b;
        a = b;
        b = c;
    }
    return b;
}

int main(void)
{
    int start_power = 10;  // Start smaller to avoid huge Fibonacci numbers
    int end_power = 20;

    printf("MatrixSize,HostMalloc_ms,DeviceMalloc_ms,MemcpyHtoD_ms,Kernel_ms,MemcpyDtoH_ms,FreeHost_ms,FreeDevice_ms,TotalMeasured_ms\n");

    for (int p = start_power; p <= end_power; p++) {
        int N = 1 << p;
        int *input, *d_input;
        unsigned long long *output, *d_output;

        // Create CUDA events
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        float host_malloc_time = 0.0f;
        float device_malloc_time = 0.0f;
        float memcpy_h2d_time = 0.0f;
        float kernel_time = 0.0f;
        float memcpy_d2h_time = 0.0f;
        float host_free_time = 0.0f;
        float device_free_time = 0.0f;

        // Host malloc timing
        hipEventRecord(start);
        input = (int*)malloc(N * sizeof(int));
        output = (unsigned long long*)malloc(N * sizeof(unsigned long long));
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&host_malloc_time, start, stop);

        // Device malloc timing
        hipEventRecord(start);
        hipMalloc(&d_input, N * sizeof(int));
        hipMalloc(&d_output, N * sizeof(unsigned long long));
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&device_malloc_time, start, stop);

        // Initialize host arrays
        for (int i = 0; i < N; i++) {
            input[i] = i % 40;  // Limit Fibonacci input size to avoid overflow
        }

        // Host to device memcpy timing
        hipEventRecord(start);
        hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&memcpy_h2d_time, start, stop);

        // Kernel execution timing
        hipEventRecord(start);
        fibonacci_kernel<<<(N + 255) / 256, 256>>>(d_input, d_output, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&kernel_time, start, stop);

        // Device to host memcpy timing
        hipEventRecord(start);
        hipMemcpy(output, d_output, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&memcpy_d2h_time, start, stop);

        // Host free timing
        hipEventRecord(start);
        free(input);
        free(output);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&host_free_time, start, stop);

        // Device free timing
        hipEventRecord(start);
        hipFree(d_input);
        hipFree(d_output);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&device_free_time, start, stop);

        // Total
        float total_measured_time = host_malloc_time + device_malloc_time + memcpy_h2d_time + kernel_time + memcpy_d2h_time + host_free_time + device_free_time;

        // Print GPU results
        printf("[GPU] %d,%.5f,%.5f,%.5f,%.5f,%.5f,%.5f,%.5f,%.5f\n", 
            N,
            host_malloc_time,
            device_malloc_time,
            memcpy_h2d_time,
            kernel_time,
            memcpy_d2h_time,
            host_free_time,
            device_free_time,
            total_measured_time
        );

        // Destroy CUDA events
        hipEventDestroy(start);
        hipEventDestroy(stop);

        // === Now benchmark CPU ===
        int *input_cpu;
        unsigned long long *output_cpu;

        double host_malloc_time_cpu = 0.0;
        double computation_time_cpu = 0.0;
        double host_free_time_cpu = 0.0;

        auto cpu_start = std::chrono::high_resolution_clock::now();
        input_cpu = (int*)malloc(N * sizeof(int));
        output_cpu = (unsigned long long*)malloc(N * sizeof(unsigned long long));
        auto cpu_stop = std::chrono::high_resolution_clock::now();
        host_malloc_time_cpu = std::chrono::duration<double, std::milli>(cpu_stop - cpu_start).count();

        for (int i = 0; i < N; i++) {
            input_cpu[i] = i
